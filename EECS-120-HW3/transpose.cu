#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;

__global__ 
void copy(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	int index = blockIdx.x * blockDim.x * blockDim.y  + threadIdx.x;
	int gridSize = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
	for(; index < N * N; index += gridSize){
		AT[index] = A[index];
	}

}

__global__ 
void matTrans2(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.x + threadIdx.y;
  int width = N;

  for (int j = 0; j < blockDim.x; j+= blockDim.y){
	if(y+j < N && x < N)
    		AT[x*width + (y+j)] = A[(y+j)*width + x];

  }
 
}

__global__ 
void matTrans(dtype* AT, dtype* A, int N)  {
	/* Fill your code here */
	__shared__ dtype tile[64][65];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.x + threadIdx.y;
	int width = N;

	for (int j = 0; j < blockDim.x; j += blockDim.y)
		if((y+j)*width + x < N * N)
			tile[threadIdx.x][threadIdx.y+ j] = A[(y+j)*width + x];

	__syncthreads();

	x = blockIdx.y * blockDim.x + threadIdx.x;  // transpose block offset
	y = blockIdx.x * blockDim.x + threadIdx.y;

	for (int j = 0; j < blockDim.x; j += blockDim.y){
		if((y+j) < N && x < N)
			AT[(y+j)*width + x] = tile[threadIdx.y + j][threadIdx.x]; //threadIdx.y=1 threadIdx.x=0 x=0 y=1
	}
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
	struct stopwatch_t* timer = NULL;
	long double t_gpu;  

	/* Setup timers */
	stopwatch_init ();
	timer = stopwatch_create ();

	int gb_x = (N + 63) / 64;
	dim3 gb(gb_x, gb_x, 1);
	dim3 tb(64, 4, 1);

	dtype* d_AT;
	dtype* d_A;
	CUDA_CHECK_ERROR (hipMalloc (&d_AT, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc (&d_A, N * N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (d_A, A, N * N * sizeof (dtype), 
				hipMemcpyHostToDevice));
	/*warm up*/
	matTrans <<<gb, tb>>> (d_AT, d_A, N);

	stopwatch_start (timer);
	/* run your kernel here */
        matTrans <<<gb, tb>>> (d_AT, d_A, N);

	hipDeviceSynchronize ();
	t_gpu = stopwatch_stop (timer);
	fprintf (stdout, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
			t_gpu, (N * N) / t_gpu * 1e-9 );
	CUDA_CHECK_ERROR (hipMemcpy (AT, d_AT, sizeof(dtype) * N * N, hipMemcpyDeviceToHost));

}

int 
main(int argc, char** argv)
{
	/* variables */
	dtype *A, *ATgpu, *ATcpu;
	int err;

	int N;

	struct stopwatch_t* timer = NULL;
	long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

	/* input and output matrices on host */
	/* output */
	ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
	ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

	/* input */
	A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

	/* Setup timers */
	stopwatch_init ();
	timer = stopwatch_create ();

	stopwatch_start (timer);
	/* compute reference array */
	cpuTranspose (A, ATcpu, N);
	t_cpu = stopwatch_stop (timer);
	fprintf (stdout, "Time to execute CPU transpose kernel: %Lg secs\n",
			t_cpu);

	/* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stdout, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

	return 0;
}
